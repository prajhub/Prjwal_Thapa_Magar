#include "hip/hip_runtime.h"
//Prajwal_Thapa_Magar_2059745



/*

This code applies a box blur to an image using a CUDA kernel function on the GPU. I
it takes in the memory locations of the input and output images, as well as their dimensions. 

*/


// Compile with:  nvcc 2059745_Blurref.cu lodepng.cpp -o cudablur
// Execute with: ./cudablur



#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>
#include "lodepng.h"


__global__ void box_blur(unsigned char * device_image_output, unsigned char * device_image_input, unsigned int width, unsigned int height)
{
	int r = 0;
	int g = 0;
	int b = 0;
	int a = 0;
	int x, y;
	int count = 0;

	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int pixel = idx * 4;

	for (x = (pixel - 4); x <= (pixel + 4); x += 4) 
	{
		if ((x > 0) && x < (height * width * 4) && ((x - 4) / (4 * width) == pixel / (4 * width))) 
		{
			for (y = (x - (4 * width)); y <=  (x + (4 * width)); y += (4 * width)) 
			{
				if (y > 0 && y < (height * width * 4)) 
				{
					r += device_image_input[y];
					g += device_image_input[1 + y];
					b += device_image_input[2 + y]; 
					count++;
				}
			}
		}
	}
	
	a = device_image_input[3 + pixel];

	device_image_output[pixel] = r / count;
	device_image_output[1 + pixel] = g / count;
	device_image_output[2 + pixel] = b / count;
	device_image_output[3 + pixel] = a;
}

int main(int argc, char **argv)
{
	unsigned int error;
	unsigned char *image;
	unsigned int width;
	unsigned int height;
	const char *input_filename = "2059745_image.png";
	const char *output_filename = "2059745_blur.png";

	error = lodepng_decode32_file(&image, &width, &height, input_filename);
	if (error) {
		printf("Error %u: %s\n", error, lodepng_error_text(error));
	}

	int array_size = width * height * 4;
	int array_bytes = array_size * sizeof(unsigned char);

	unsigned char host_image_input[array_size * 4];
	unsigned char host_image_output[array_size * 4];

	for (int i = 0; i < array_size; i++) {
		host_image_input[i] = image[i];
	}

	// Declaring the device's memory pointers.
	unsigned char * d_in;
	unsigned char * d_out;

	// Allocating device's  memory.
	hipMalloc((void**) &d_in, array_bytes);
	hipMalloc((void**) &d_out, array_bytes);

	// Transferring the host image input data to device memory..
	hipMemcpy(d_in, host_image_input, array_bytes, hipMemcpyHostToDevice);

	// Invoking the kernel function.
	box_blur<<<height, width>>>(d_out, d_in, width, height);

	// Transferring the processed data from the device memory back to the host memory.
	hipMemcpy(host_image_output, d_out, array_bytes, hipMemcpyDeviceToHost);
	
	error = lodepng_encode32_file(output_filename, host_image_output, width, height);
	if (error) {
		printf("error %u: %s\n", error, lodepng_error_text(error));
	}

	printf("\n Blurred image is saved on the same directory with name 2059745_blur\n\n");

	// deallocating the device memory
	hipFree(d_in);
	hipFree(d_out);

	return 0;
}