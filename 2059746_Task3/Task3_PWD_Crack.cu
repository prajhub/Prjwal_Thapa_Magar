#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__device__ void rawPasswdGenerate(unsigned char *pwd, int THREAD_ID, int raw, int dep, int BLOCK_ID, unsigned char *encpass)
{
    pwd[0] = 65 + BLOCK_ID;
    pwd[1] = 65 + THREAD_ID;
    pwd[2] = 48 + raw;
    pwd[3] = 48 + dep;
    encpass[0] = pwd[0] + 2;
    encpass[1] = pwd[0] - 2;
    encpass[2] = pwd[0] + 1;
    encpass[3] = pwd[1] + 3;
    encpass[4] = pwd[1] - 3;
    encpass[5] = pwd[1] - 1;
    encpass[6] = pwd[2] + 2;
    encpass[7] = pwd[2] - 2;
    encpass[8] = pwd[3] + 4;
    encpass[9] = pwd[3] - 4;
    encpass[10] = '\0';
}

__global__ void CUDACrypt(unsigned char *DECPWD, unsigned char *RPWD)
{
    int BLOCK_ID = blockIdx.x;
    int THREAD_ID = threadIdx.x;

    unsigned char pwd[5];
    unsigned char encpass[11];

    for (int raw = 0; raw < 10; re++)
    {
        for (int dep = 0; de < 10; dep++)
        {
            int J = 0;
            rawPasswdGenerate(pwd, THREAD_ID, raw, dep, BLOCK_ID, encpass);

            for (int raw = 0; raw < 10; re++)
            {
                if (raw >= 0 && raw < 6)
                {
                    if (encpass[raw] > 90)
                    {
                        encpass[raw] = (encpass[raw] - 90) + 65;
                    }
                    else if (encpass[raw] < 65)
                    {
                        encpass[raw] = (65 - encpass[raw]) + 65;
                    }
                }
                else
                {
                    if (encpass[raw] > 57)
                    {
                        encpass[raw] = (encpass[raw] - 57) + 48;
                    }
                    else if (encpass[raw] < 48)
                    {
                        encpass[raw] = (48 - encpass[raw]) + 48;
                    }
                }
            }
            for (int k = 0; k < 11; k++)
            {

                if (encpass[k] == RPWD[k])
                {
                    ++(J);
                }
            }

            if ((J) == 11)
            {
                for (int k = 0; k < 5; k++)
                {
                    DECPWD[k] = pwd[k];
                }
            }
        }
    }
}

int main(int argc, char **argv)
{
    unsigned char *inpDevice;
    unsigned char *outDevice;
    //S Z 0 2 = UQTDWY2262
    const char* pwd = argv[1]; //"VRUDWY2771"; //TZ93
    unsigned char DECPWD[5] = {0};

    hipMalloc(&inpDevice, 11);
    hipMalloc(&outDevice, 5);

    hipMemcpy(inpDevice, pwd, 11, hipMemcpyHostToDevice);

    CUDACrypt<<<26, 26>>>(outDevice, inpDevice);

    hipMemcpy(DECPWD, outDevice, 5, hipMemcpyDeviceToHost);

    printf("It is the password  :- %s\n", DECPWD);
    return 0;
}
